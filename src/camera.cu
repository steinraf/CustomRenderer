#include "hip/hip_runtime.h"
//
// Created by steinraf on 21/08/22.
//

#include "camera.h"
#include "utility/warp.h"

__device__ __host__ Camera::Camera(Vector3f origin, Vector3f lookAt, Vector3f _up, float vFOV,
                                   float aspectRatio, float aperture, float focusDist)
        : origin(origin), lensRadius(aperture / 2.0f){





    constexpr int noriConvert = 1; // -1 for nori, 1 for correct handedness


    front = (lookAt - origin).normalized();
    right = noriConvert*(_up.cross(-front)).normalized();
    up = front.cross(noriConvert*-right);

    const float halfHeight = tan(vFOV * M_PIf * 0.5f / 180.0f);
    const float halfWidth = aspectRatio * halfHeight;

    const Vector3f halfU = halfWidth * focusDist * right;
    const Vector3f halfV = halfHeight * focusDist * up;

    upperLeft = origin
                - halfU
                + halfV
                + focusDist * front;


    horizontal = 2.0f * halfU;
    vertical = -2.0f * halfV;
}

__device__ Ray3f Camera::getRay(float s, float t, Sampler &sampler) const{

    //sample = ((0.5x, -0.5*aspect*y, 1z) + (1.0, -1.0f/aspect, 0.f) * perspective).inverse()


    const Vector2f randomDisk = lensRadius * Warp::squareToUniformDisk(sampler.getSample2D());
    const Vector3f offset = right * randomDisk[0] + up * randomDisk[1];

    const Vector3f pos = origin + offset;


    return {pos, upperLeft + s * horizontal + t * vertical - pos};
}