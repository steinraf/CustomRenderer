#include "hip/hip_runtime.h"
//
// Created by steinraf on 28/11/22.
//

#include "../acceleration/bvh.h"
#include "../shapes/triangle.h"
#include "areaLight.h"

//__host__ __device__ AreaLight::AreaLight(const Color3f &radiance) noexcept
//    :radiance(radiance), blas(nullptr){
////        assert(blas);
////        printf("Initialized Area Light with radiance (%f, %f, %f)\n", blas->radiance[0], blas->radiance[1], blas->radiance[2]);
////        printf("THIS: %p\n", this);
//}

__device__ float AreaLight::pdf(const EmitterQueryRecord &emitterQueryRecord) const noexcept {
    assert(blas);
    ShapeQueryRecord sRec{
            emitterQueryRecord.ref,
            emitterQueryRecord.p};

    //TODO take into account feedback received in exercise

    return (emitterQueryRecord.ref - emitterQueryRecord.p).squaredNorm() * blas->pdfSurface(sRec) / abs(emitterQueryRecord.n.dot(-emitterQueryRecord.wi) + EPSILON);
}

__device__ Color3f AreaLight::sample(EmitterQueryRecord &emitterQueryRecord, const Vector2f &sample) const noexcept {

    //    printf("AreaEmitter BLAS NumPrimitives is %lu\n", blas->numPrimitives);


    assert(isEmitter());


    ShapeQueryRecord sRec{
            emitterQueryRecord.ref};

    assert(blas);
    blas->sampleSurface(sRec, sample);

    //    printf("Sampled surface\n");

    //    assert(emitterQueryRecord.p != emitterQueryRecord.ref);

    //    printf("p(%f, %f, %f), ref(%f, %f, %f)\n",
    //           sRec.p[0], sRec.p[1], sRec.p[2],
    //           emitterQueryRecord.ref[0], emitterQueryRecord.ref[1], emitterQueryRecord.ref[2]
    //    );


    emitterQueryRecord.p = sRec.p;
    emitterQueryRecord.wi = (emitterQueryRecord.p - emitterQueryRecord.ref).normalized();
    emitterQueryRecord.shadowRay = {
            emitterQueryRecord.ref,
            emitterQueryRecord.wi,
            EPSILON,
            (emitterQueryRecord.p - emitterQueryRecord.ref).norm() - EPSILON};


    emitterQueryRecord.n = sRec.n.normalized();
    emitterQueryRecord.pdf = pdf(emitterQueryRecord);

    return eval(emitterQueryRecord) / emitterQueryRecord.pdf;
}

//__device__ Color3f AreaLight::eval(const EmitterQueryRecord &emitterQueryRecord) const noexcept
