#include "hip/hip_runtime.h"
//
// Created by steinraf on 24/10/22.
//

#include "meshLoader.h"
#include "vector.h"
#include "../cudaHelpers.h"

HostMeshInfo loadMesh(const std::filesystem::path &filePath){

    std::cout << "Reading mesh " + filePath.filename().string() + " ...\n";
    std::ifstream file(filePath, std::ios::in);
    if(!file.is_open())
        throw std::runtime_error("Error, file " + filePath.string() + " could not be opened in the MeshLoader.");

    std::string lineString;

    thrust::host_vector<Vector3f> vertices;
    thrust::host_vector<Vector2f> textures;
    thrust::host_vector<Vector3f> normals;


    thrust::host_vector<int> vertexIndices1;
    thrust::host_vector<int> vertexIndices2;
    thrust::host_vector<int> vertexIndices3;

    thrust::host_vector<int> textureIndices1;
    thrust::host_vector<int> textureIndices2;
    thrust::host_vector<int> textureIndices3;

    thrust::host_vector<int> normalIndices1;
    thrust::host_vector<int> normalIndices2;
    thrust::host_vector<int> normalIndices3;


//    thrust::host_vector<FaceElement> faces;

    while(std::getline(file, lineString)){
        std::istringstream line{lineString};
        std::string start;

        line >> start;

//        std::cout << "Read line: " << lineString << '\n';

        if(start == "v"){
            float x, y, z;
            line >> x >> y >> z;
            vertices.push_back({x, y, z});

        }else if(start == "vt"){
            float u, v, w;
            line >> u >> v >> w;
            textures.push_back({u, v});
        }else if(start == "vn"){
            float x, y, z;
            line >> x >> y >> z;
            normals.push_back({x, y, z});

        }else if(start == "f"){
            std::string e1, e2, e3, e4;
            line >> e1 >> e2 >> e3 >> e4;

            std::istringstream s1(e1), s2(e2), s3(e3);

            int v1, v2, v3,
                    t1, t2, t3,
                    n1, n2, n3;

            char delim;

            s1 >> v1 >> delim >> t1 >> delim >> n1;
            s2 >> v2 >> delim >> t2 >> delim >> n2;
            s3 >> v3 >> delim >> t3 >> delim >> n3;

            vertexIndices1.push_back(v1 - 1);
            vertexIndices2.push_back(v2 - 1);
            vertexIndices3.push_back(v3 - 1);

            textureIndices1.push_back(t1 - 1);
            textureIndices2.push_back(t2 - 1);
            textureIndices3.push_back(t3 - 1);

            normalIndices1.push_back(n1 - 1);
            normalIndices2.push_back(n2 - 1);
            normalIndices3.push_back(n3 - 1);

            if(!e4.empty()){
                int v4, t4, n4;
                std::istringstream s4(e4);
                s4 >> v4 >> delim >> t4 >> delim >> n4;

                vertexIndices1.push_back(v1 - 1);
                vertexIndices2.push_back(v3 - 1);
                vertexIndices3.push_back(v4 - 1);

                textureIndices1.push_back(t1 - 1);
                textureIndices2.push_back(t3 - 1);
                textureIndices3.push_back(t4 - 1);

                normalIndices1.push_back(n1 - 1);
                normalIndices2.push_back(n3 - 1);
                normalIndices3.push_back(n4 - 1);
            }
        }
    }

    return {
            vertices,
            textures,
            normals,
            {vertexIndices1, vertexIndices2, vertexIndices3},
            {textureIndices1, textureIndices2, textureIndices3},
            {normalIndices1, normalIndices2, normalIndices3}
    };
}


DeviceMeshInfo meshToGPU(const HostMeshInfo &mesh) noexcept {
    const auto numTriangles = mesh.normalsIndices.first.size();

    std::vector<Triangle> hostTriangles(numTriangles);

    const auto &[
            vertices,
            textures,
            normals,
            vertexIndexList,
            textureIndexList,
            normalIndexList] = mesh;

#pragma omp parallel for
    for(int i = 0; i < numTriangles; ++i){
        hostTriangles[i] = {
                vertices[vertexIndexList.first[i]],
                vertices[vertexIndexList.second[i]],
                vertices[vertexIndexList.third[i]],
                textures[textureIndexList.first[i]],
                textures[textureIndexList.second[i]],
                textures[textureIndexList.third[i]],
                normals[normalIndexList.first[i]],
                normals[normalIndexList.second[i]],
                normals[normalIndexList.third[i]],
                BSDF{Material::DIFFUSE}
        };
    }

    thrust::device_vector<Triangle> deviceTriangles(hostTriangles);


    TriaToAABB triangleToAABB;
    AABB aabb{};
    AABBAdder aabbAddition;


    AABB maxBoundingBox = thrust::transform_reduce(deviceTriangles.begin(), deviceTriangles.end(),
                                                   triangleToAABB, aabb, aabbAddition);

    //TODO maybe try to only compute total BB once (but morton codes require normalize with BB)

//    printf("Bounding box of total scene is (%f, %f, %f), (%f, %f, %f)\n",
//            maxBoundingBox.min[0], maxBoundingBox.min[1], maxBoundingBox.min[2],
//           maxBoundingBox.max[0], maxBoundingBox.max[1], maxBoundingBox.max[2]
//    );

    TriangleToMortonCode triangleToMortonCode(maxBoundingBox);

    thrust::device_vector<uint32_t> mortonCodes(numTriangles);

    thrust::transform(deviceTriangles.begin(), deviceTriangles.end(), mortonCodes.begin(),
                      triangleToMortonCode);

    thrust::sort_by_key(mortonCodes.begin(), mortonCodes.end(), deviceTriangles.begin());
    //TODO maybe use radix sort instead of default sort

    return {deviceTriangles, mortonCodes};
}
