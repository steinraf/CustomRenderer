#include "hip/hip_runtime.h"
//
// Created by steinraf on 24/10/22.
//

#include "meshLoader.h"
#include "vector.h"
#include "../cudaHelpers.h"
#include <thrust/transform_scan.h>
HostMeshInfo loadMesh(const std::filesystem::path &filePath, const Affine3f &transform) noexcept(false){

    std::cout << "Reading mesh " + filePath.filename().string() + " ...\n";
    std::ifstream file(filePath, std::ios::in);
    if(!file.is_open())
        throw std::runtime_error("Error, file " + filePath.string() + " could not be opened in the MeshLoader.");

    std::string lineString;

    thrust::host_vector<Vector3f> vertices;
    thrust::host_vector<Vector2f> textures;
    thrust::host_vector<Vector3f> normals;


    thrust::host_vector<int> vertexIndices1;
    thrust::host_vector<int> vertexIndices2;
    thrust::host_vector<int> vertexIndices3;

    thrust::host_vector<int> textureIndices1;
    thrust::host_vector<int> textureIndices2;
    thrust::host_vector<int> textureIndices3;

    thrust::host_vector<int> normalIndices1;
    thrust::host_vector<int> normalIndices2;
    thrust::host_vector<int> normalIndices3;


    while(std::getline(file, lineString)){
        std::istringstream line{lineString};
        std::string start;

        line >> start;

        if(start == "v"){
            float x, y, z;
            line >> x >> y >> z;
            vertices.push_back(Vector3f{x, y, z}.applyTransform(transform));
//            std::cout << filePath << ", found coord x y z " << x << ' ' << y << ' ' << z << '\n';


        }else if(start == "vt"){
            float u, v, w;
            line >> u >> v >> w;
            textures.push_back({u, v});
//            std::cout << filePath << ", found texture u v w " << u << ' ' << v << ' ' << w << '\n';
        }else if(start == "vn"){
            float x, y, z;
            line >> x >> y >> z;
            normals.push_back(Vector3f{x, y, z}.applyTransform(transform, true));

        }else if(start == "f"){
            std::string e1, e2, e3, e4;
            line >> e1 >> e2 >> e3 >> e4;

            std::istringstream s1(e1), s2(e2), s3(e3);

            int v1, v2, v3,
                t1, t2, t3,
                n1, n2, n3;

            char delim;

            s1 >> v1 >> delim >> t1 >> delim >> n1;
            s2 >> v2 >> delim >> t2 >> delim >> n2;
            s3 >> v3 >> delim >> t3 >> delim >> n3;

            vertexIndices1.push_back(v1 - 1);
            vertexIndices2.push_back(v2 - 1);
            vertexIndices3.push_back(v3 - 1);

            textureIndices1.push_back(t1 - 1);
            textureIndices2.push_back(t2 - 1);
            textureIndices3.push_back(t3 - 1);

            normalIndices1.push_back(n1 - 1);
            normalIndices2.push_back(n2 - 1);
            normalIndices3.push_back(n3 - 1);

            if(!e4.empty()){
                int v4, t4, n4;
                std::istringstream s4(e4);
                s4 >> v4 >> delim >> t4 >> delim >> n4;

                vertexIndices3.push_back(v4 - 1);
                vertexIndices1.push_back(v1 - 1);
                vertexIndices2.push_back(v3 - 1);


                textureIndices3.push_back(t4 - 1);
                textureIndices1.push_back(t1 - 1);
                textureIndices2.push_back(t3 - 1);


                normalIndices3.push_back(n4 - 1);
                normalIndices1.push_back(n1 - 1);
                normalIndices2.push_back(n3 - 1);

            }
        }
    }

    return {
            vertices,
            textures,
            normals,
            {vertexIndices1, vertexIndices2, vertexIndices3},
            {textureIndices1, textureIndices2, textureIndices3},
            {normalIndices1, normalIndices2, normalIndices3}
    };
}


DeviceMeshInfo meshToGPU(const HostMeshInfo &mesh) noexcept {
    const auto numTriangles = mesh.normalsIndices.first.size();

    std::vector<Triangle> hostTriangles(numTriangles);

    const auto &[
            vertices,
            textures,
            normals,
            vertexIndexList,
            textureIndexList,
            normalIndexList] = mesh;

#pragma omp parallel for
    for(int i = 0; i < numTriangles; ++i){
        hostTriangles[i] = {
                vertices[vertexIndexList.first[i]],
                vertices[vertexIndexList.second[i]],
                vertices[vertexIndexList.third[i]],
                textures[textureIndexList.first[i]],
                textures[textureIndexList.second[i]],
                textures[textureIndexList.third[i]],
                normals[normalIndexList.first[i]],
                normals[normalIndexList.second[i]],
                normals[normalIndexList.third[i]],
                BSDF{Material::DIFFUSE}
        };
    }

    thrust::device_vector<Triangle> deviceTriangles(hostTriangles);


    TriaToAABB triangleToAABB;
    AABB aabb{};
    AABBAdder aabbAddition;

    TriaToArea triangleToArea;


    AABB maxBoundingBox = thrust::transform_reduce(deviceTriangles.begin(), deviceTriangles.end(),
                                                   triangleToAABB, aabb, aabbAddition);

    float totalTriaArea = thrust::transform_reduce(deviceTriangles.begin(), deviceTriangles.end(),
                                                  triangleToArea, 0.f, thrust::plus<float>());

    TriangleToCDF triangleToCdf(totalTriaArea);
    thrust::device_vector<float> areaCDF(numTriangles);

    thrust::transform_inclusive_scan(deviceTriangles.begin(), deviceTriangles.end(),
                                     areaCDF.begin(), triangleToCdf, thrust::plus<float>());

    printf("\tTotal area of all triangles is %f\n", totalTriaArea);

    //TODO maybe try to only compute total BB once (but morton codes require normalize with BB)

//    printf("Bounding box of total scene is (%f, %f, %f), (%f, %f, %f)\n",
//            maxBoundingBox.min[0], maxBoundingBox.min[1], maxBoundingBox.min[2],
//           maxBoundingBox.max[0], maxBoundingBox.max[1], maxBoundingBox.max[2]
//    );

    TriangleToMortonCode triangleToMortonCode(maxBoundingBox);

    thrust::device_vector<uint32_t> mortonCodes(numTriangles);

    thrust::transform(deviceTriangles.begin(), deviceTriangles.end(), mortonCodes.begin(),
                      triangleToMortonCode);

    thrust::sort_by_key(mortonCodes.begin(), mortonCodes.end(), deviceTriangles.begin());
    //TODO maybe use radix sort instead of default sort

    return {deviceTriangles, mortonCodes, areaCDF, totalTriaArea};
}
