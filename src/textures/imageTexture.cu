//
// Created by steinraf on 02/12/22.
//

#include "../cudaHelpers.h"
#include "imageTexture.h"
#include <thrust/device_vector.h>
#include <thrust/transform_scan.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"


__host__ Texture::Texture(const std::filesystem::path &imagePath, bool isEnvMap) noexcept {
    assert(!imagePath.string().empty());
    width = height = dim = 0;
    printf("Loading texture %s\n", imagePath.c_str());
    auto *hostTexture = (Vector3f *) stbi_loadf(imagePath.c_str(), &width, &height, &dim, 3);

#ifndef NDEBUG
    if(!hostTexture){
        printf("The failure reason is %s\n", stbi__g_failure_reason);
        assert(false);
    }
#endif

    assert(hostTexture);


    printf("Size of the image is %i / %i\n", width, height);


    checkCudaErrors(hipMalloc(&deviceTexture, width * height * sizeof(Vector3f)));
    checkCudaErrors(hipMalloc(&deviceCDF, width * height * sizeof(float)));

    checkCudaErrors(hipMemcpy(deviceTexture, hostTexture, width * height * sizeof(Vector3f), hipMemcpyHostToDevice));


    ColorToRadiance colorToRadiance(deviceTexture, width, height, isEnvMap);


    thrust::device_ptr<Vector3f> deviceTexturePtr{deviceTexture};
    float totalSum = thrust::transform_reduce(deviceTexturePtr, deviceTexturePtr + width*height,
                                                    colorToRadiance, 0.f, thrust::plus<float>());

    ColorToCDF colorToCdf{deviceTexture, width, height, totalSum, isEnvMap};

    thrust::transform_inclusive_scan(deviceTexturePtr, deviceTexturePtr + width*height,
                                     deviceCDF, colorToCdf, thrust::plus<float>());

    assert(dim == 3);
}