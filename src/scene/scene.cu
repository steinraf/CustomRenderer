#include "hip/hip_runtime.h"
//
// Created by steinraf on 19/08/22.
//

#include "scene.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


#include <fstream>
#include <thread>

__host__ Scene::Scene(SceneRepresentation &&sceneRepr, Device dev) : sceneRepresentation(sceneRepr),
                                                                     imageBufferByteSize(sceneRepr.sceneInfo.width * sceneRepr.sceneInfo.height * sizeof(Vector3f)),
                                                                     blockSize(sceneRepr.sceneInfo.width / blockSizeX + 1, sceneRepr.sceneInfo.height / blockSizeY + 1),
                                                                     device(dev),
                                                                     hostDeviceMeshTriangleVec(sceneRepresentation.meshInfos.size()),
                                                                     hostDeviceMeshCDF(sceneRepresentation.meshInfos.size()),
                                                                     totalMeshArea(sceneRepresentation.meshInfos.size()),
                                                                     hostDeviceEmitterTriangleVec(sceneRepresentation.emitterInfos.size()),
                                                                     hostDeviceEmitterCDF(sceneRepresentation.emitterInfos.size()),
                                                                     totalEmitterArea(sceneRepresentation.emitterInfos.size()),
                                                                     deviceCamera(sceneRepr.cameraInfo.origin,
                                                                                  sceneRepr.cameraInfo.target,
                                                                                  sceneRepr.cameraInfo.up,
                                                                                  sceneRepr.cameraInfo.fov,
                                                                                  static_cast<float>(sceneRepr.sceneInfo.width) / static_cast<float>(sceneRepr.sceneInfo.height),
                                                                                  sceneRepr.cameraInfo.aperture,
                                                                                  sceneRepr.cameraInfo.focusDist) {//(customRenderer::getCameraOrigin() - customRenderer::getCameraLookAt()).norm()){


    if(dev == CPU) {
        checkCudaErrors(hipMalloc(&deviceImageBuffer, imageBufferByteSize));
        checkCudaErrors(hipMalloc(&deviceImageBufferDenoised, imageBufferByteSize));
    } else {

        //        checkCudaErrors(hipMalloc(&deviceImageBuffer, imageBufferByteSize)); Allocated by OpenGL instead
        checkCudaErrors(hipMalloc(&deviceImageBufferDenoised, imageBufferByteSize));
        initOpenGL();
    }

    checkCudaErrors(hipMalloc(&deviceFeatureBuffer, sizeof(FeatureBuffer) * sceneRepresentation.sceneInfo.width *
                                                             sceneRepresentation.sceneInfo.height));


    checkCudaErrors(hipMalloc(&deviceCurandState,
                               sceneRepresentation.sceneInfo.width * sceneRepresentation.sceneInfo.height *
                                       sizeof(hiprandState)));

    cudaHelpers::initRng<<<blockSize, threadSize>>>(sceneRepresentation.sceneInfo.width, sceneRepresentation.sceneInfo.height, deviceCurandState);
    checkCudaErrors(hipGetLastError());


    checkCudaErrors(hipMalloc(&meshAccelerationStructure, sizeof(TLAS)));
//    checkCudaErrors(hipMalloc(&emitterAccelerationStructure, sizeof(TLAS)));

    // No need to sync because can run independently
    //    checkCudaErrors(hipDeviceSynchronize());

    auto numMeshes = sceneRepresentation.meshInfos.size();


    std::vector<BLAS *> hostMeshBlasVector(numMeshes);


    clock_t meshLoadStart = clock();
#pragma omp parallel for
    for(size_t i = 0; i < numMeshes; ++i) {
        hostMeshBlasVector[i] = getMeshFromFile(sceneRepr.meshInfos[i].filename,
                                                hostDeviceMeshTriangleVec[i],
                                                hostDeviceMeshCDF[i],
                                                totalMeshArea[i],
                                                sceneRepr.meshInfos[i].transform,
                                                sceneRepr.meshInfos[i].bsdf,
                                                sceneRepr.meshInfos[i].normalMap);
    }

    auto numEmitters = sceneRepresentation.emitterInfos.size();

    std::vector<BLAS *> hostEmitterBlasVector(numEmitters);

    std::vector<AreaLight> hostAreaLights(numEmitters);
    for(size_t i = 0; i < numEmitters; ++i) {
        hostAreaLights[i] = AreaLight(sceneRepr.emitterInfos[i].radiance);
    }

    AreaLight *deviceAreaLights;
    checkCudaErrors(hipMalloc(&deviceAreaLights, sizeof(AreaLight) * numEmitters));
    checkCudaErrors(hipMemcpy(deviceAreaLights, hostAreaLights.data(), sizeof(AreaLight) * numEmitters,
                               hipMemcpyHostToDevice));


#pragma omp parallel for
    for(size_t i = 0; i < numEmitters; ++i) {
        hostEmitterBlasVector[i] = getMeshFromFile(sceneRepr.emitterInfos[i].filename,
                                                   hostDeviceEmitterTriangleVec[i],
                                                   hostDeviceEmitterCDF[i],
                                                   totalEmitterArea[i],
                                                   sceneRepr.emitterInfos[i].transform,
                                                   sceneRepr.emitterInfos[i].bsdf,
                                                   sceneRepr.emitterInfos[i].normalMap,
                                                   deviceAreaLights + i);
    }


    std::cout << "Loading all Geometry took "
              << ((double) (clock() - meshLoadStart)) / CLOCKS_PER_SEC
              << " seconds.\n";


    BLAS **deviceBlasArr = cudaHelpers::hostVecToDeviceRawPtr(hostMeshBlasVector);
    BLAS **deviceEmitterBlasArr = cudaHelpers::hostVecToDeviceRawPtr(hostEmitterBlasVector);

    cudaHelpers::constructTLAS<<<1, 1>>>(meshAccelerationStructure,
                                         deviceBlasArr, numMeshes,
                                         deviceEmitterBlasArr, numEmitters,
                                         EnvironmentEmitter{sceneRepresentation.environmentInfo.texture});

    checkCudaErrors(hipGetLastError());

    hostImageBuffer = new Vector3f[imageBufferByteSize];
    hostImageBufferDenoised = new Vector3f[imageBufferByteSize];
}

__host__ Scene::~Scene() {


    delete[] hostImageBuffer;
    delete[] hostImageBufferDenoised;

    if(device == CPU) {
        checkCudaErrors(hipDeviceSynchronize());
        //                checkCudaErrors(hipFree(deviceImageBuffer));
        //        glDeleteVertexArrays(1, &VAO);
        //        glDeleteBuffers(1, &VBO);
        //        glDeleteBuffers(1, &EBO);
    } else {
        hipGraphicsUnmapResources(1, &cudaPBOResource, nullptr);
    }
    //    glfwTerminate();
    cudaHelpers::freeVariables<<<blockSize, threadSize>>>();
}


void Scene::render() {

    volatile bool currentlyRendering = true;

    std::thread drawingThread;

    if(device == GPU) {
        drawingThread = std::thread{[this](Vector3f *v, volatile bool &render) {
                                        OpenGLDraw(v, render);
                                    },
                                    deviceImageBuffer, std::ref(currentlyRendering)};
    }


    std::cout << "Starting render...\n";

    unsigned *deviceCounter;

    checkCudaErrors(hipMalloc(&deviceCounter, sizeof(unsigned)));
    checkCudaErrors(hipMemset(deviceCounter, 0, sizeof(unsigned)));

    std::cout << "Starting draw thread...\n";


    cudaHelpers::render<<<blockSize, threadSize>>>(deviceImageBuffer, deviceCamera, meshAccelerationStructure,
                                                   sceneRepresentation.sceneInfo.width, sceneRepresentation.sceneInfo.height, sceneRepresentation.sceneInfo.samplePerPixel, sceneRepresentation.sceneInfo.maxRayDepth,
                                                   deviceCurandState, deviceFeatureBuffer, deviceCounter);
    checkCudaErrors(hipGetLastError());


    std::cout << "Synchronizing GPU...\n";
    checkCudaErrors(hipDeviceSynchronize());


    std::cout << "Starting denoise...\n";
    checkCudaErrors(hipMemcpy(hostImageBuffer, deviceImageBuffer, imageBufferByteSize, hipMemcpyDeviceToHost));
    float *deviceWeights;
    checkCudaErrors(hipMalloc(&deviceWeights, sceneRepresentation.sceneInfo.width * sceneRepresentation.sceneInfo.height * sizeof(float)));
    checkCudaErrors(hipMemset(deviceWeights, 0.f, sceneRepresentation.sceneInfo.width * sceneRepresentation.sceneInfo.height * sizeof(float)));

    cudaHelpers::denoise<<<blockSize, threadSize>>>(deviceImageBuffer, deviceImageBufferDenoised, deviceFeatureBuffer, deviceWeights, sceneRepresentation.sceneInfo.width, sceneRepresentation.sceneInfo.height, sceneRepresentation.cameraInfo.origin);
    checkCudaErrors(hipDeviceSynchronize());
    cudaHelpers::denoiseApplyWeights<<<blockSize, threadSize>>>(deviceImageBufferDenoised, deviceWeights, sceneRepresentation.sceneInfo.width, sceneRepresentation.sceneInfo.height);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(deviceWeights));

    checkCudaErrors(
            hipMemcpy(hostImageBufferDenoised, deviceImageBufferDenoised, imageBufferByteSize,
                       hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());


    currentlyRendering = false;

    if(device == GPU) {
        drawingThread.join();
    }


    std::cout << "Joined draw thread...\n";
}

__host__ void Scene::renderGPU() {


    for(int i = 0; i < 10000; ++i) {
        std::cout << "Rendering frame " << i << '\n';
        render();
    }
}

__host__ void Scene::renderCPU() {
    //    initOpenGL();

    clock_t start, stop;
    start = clock();

    render();

    stop = clock();
    double timer_seconds = ((double) (stop - start)) / CLOCKS_PER_SEC;
    std::cout << "Computation took " << timer_seconds << " seconds.\n";

    std::cout << "Writing resulting image to disk...\n";

    if(!std::filesystem::exists("./data"))
        std::filesystem::create_directory("./data");

    const std::string pngPath = "./data/image.png";
    const std::string pngPathDenoised = "./data/imageDenoised.png";

    const std::string hdrPath = "./data/image.hdr";
    const std::string hdrPathDenoised = "./data/imageDenoised.hdr";

    //    stbi_set_flip_vertically_on_load(true);

    const bool didHDR = stbi_write_hdr(hdrPath.c_str(), sceneRepresentation.sceneInfo.width,
                                       sceneRepresentation.sceneInfo.height, 3, (float *) hostImageBuffer);
    assert(didHDR);

    const bool didHDRDenoised = stbi_write_hdr(hdrPathDenoised.c_str(), sceneRepresentation.sceneInfo.width,
                                       sceneRepresentation.sceneInfo.height, 3, (float *) hostImageBufferDenoised);
    assert(didHDRDenoised);


    pngwriter png(sceneRepresentation.sceneInfo.width, sceneRepresentation.sceneInfo.height, 1., pngPath.c_str());
    pngwriter pngDenoised(sceneRepresentation.sceneInfo.width, sceneRepresentation.sceneInfo.height, 1.,
                          pngPathDenoised.c_str());

    auto gammaCorrect = [](float value) {
        if(value <= 0.0031308f) return std::clamp(12.92f * value, 0.f, 1.f);
        return std::clamp(1.055f * std::pow(value, 1.f / 2.4f) - 0.055f, 0.f, 1.f);
    };


#pragma omp parallel for
    for(int j = 0; j < sceneRepresentation.sceneInfo.height; j++) {
        for(int i = 0; i < sceneRepresentation.sceneInfo.width; i++) {
            const int idx = j * sceneRepresentation.sceneInfo.width + i;
            png.plot(i + 1, sceneRepresentation.sceneInfo.height - j,
                     gammaCorrect(hostImageBuffer[idx][0]),
                     gammaCorrect(hostImageBuffer[idx][1]),
                     gammaCorrect(hostImageBuffer[idx][2]));
            pngDenoised.plot(i + 1, sceneRepresentation.sceneInfo.height - j,
                             gammaCorrect(hostImageBufferDenoised[idx][0]),
                             gammaCorrect(hostImageBufferDenoised[idx][1]),
                             gammaCorrect(hostImageBufferDenoised[idx][2]));
        }
    }

    png.close();
    pngDenoised.close();
}

__host__ void Scene::initOpenGL() {
    assert(false);
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    window = glfwCreateWindow(sceneRepresentation.sceneInfo.width, sceneRepresentation.sceneInfo.height, "Raytracing",
                              nullptr, nullptr);
    if(!window) {
        std::cerr << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        throw std::runtime_error("GLFW WINDOW ERROR");
    }

    glfwMakeContextCurrent(window);
    //    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    //    glfwSetCursorPosCallback(window, mouse_callback);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);

    glfwSwapInterval(1);

    if(!gladLoadGLLoader((GLADloadproc) glfwGetProcAddress)) {
        std::cerr << "Failed to initialize GLAD" << std::endl;
        throw std::runtime_error("GLAD INIT ERROR");
    }

    loadShader();

    glEnable(GL_DEPTH_TEST);


    switch(device) {
        break;
        case GPU:

            glGenBuffers(1, &PBO);
            glBindBuffer(GL_PIXEL_UNPACK_BUFFER, PBO);
            glBufferData(GL_PIXEL_UNPACK_BUFFER,
                         sizeof(Vector3f) * sceneRepresentation.sceneInfo.width * sceneRepresentation.sceneInfo.height,
                         nullptr, GL_STREAM_DRAW);
            glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

            //    glGenTextures(1, &tex);
            //    glBindTexture(GL_TEXTURE_2D, tex);
            //    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

            checkCudaErrors(hipGraphicsGLRegisterBuffer(&cudaPBOResource, PBO, hipGraphicsRegisterFlagsNone));

            checkCudaErrors(hipGraphicsMapResources(1, &cudaPBOResource, nullptr));

            checkCudaErrors(
                    hipGraphicsResourceGetMappedPointer((void **) &deviceImageBuffer,
                                                         const_cast<size_t *>(&imageBufferByteSize), cudaPBOResource));

            checkCudaErrors(hipDeviceSynchronize());
            break;
        case CPU:
            glGenVertexArrays(1, &VAO);
            glGenBuffers(1, &VBO);

            //            glGenBuffers(1, &EBO);
    }
}

__host__ void Scene::OpenGLDraw(Vector3f *deviceVector, volatile bool &isRendering) {

    glGenVertexArrays(1, &VAO);

    std::cout << "Starting OpenGLDraw\n";

    float vertices[] = {
            -0.5f, -0.5f, 0.0f,
            0.5f, -0.5f, 0.0f,
            0.0f, 0.5f, 0.0f};

    std::cout << "Hehe 2";

    //    float *vertices;

    glBindVertexArray(VAO);

    std::cout << "Hehe 2.5";

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    std::cout << "Hehe 3";

    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0 * 3 * sizeof(float), nullptr);

    std::cout << "Hehe 4";

    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glBindVertexArray(0);

    std::cout << "Hehe 1";
    while(isRendering) {
        //        glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);


        glUseProgram(shaderID);
        glBindVertexArray(VAO);
        //        checkCudaErrors(hipMemcpy(hostImageBuffer, deviceImageBuffer, imageBufferByteSize, hipMemcpyDeviceToHost));

        //        std::cout << deviceImageBuffer[0] << '\n';
        //        glBufferData(GL_ARRAY_BUFFER, 2 * width * height * sizeof(Vector3f), NULL, GL_STATIC_DRAW);
        //
        //        glBufferSubData(GL_ARRAY_BUFFER, 0, width * height * sizeof(Vector3f), hostImageBuffer);
        //        glBufferSubData(GL_ARRAY_BUFFER, width * height * sizeof(Vector3f), width * height * sizeof(Vector3f), hostCoordinateVector);

        //        checkCudaErrors(hipDeviceSynchronize());
        //        glBufferData(GL_ARRAY_BUFFER, width * height * sizeof(Vector3f), hostImageBuffer, GL_STATIC_DRAW);


        //            glDrawArrays(GL_POINTS, 0, width*height);

        //        glEnable(GL_PROGRAM_POINT_SIZE);


        glDrawArrays(GL_TRIANGLES, 0, sizeof(vertices) / 3);


        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    std::cout << "Exiting :sadge:\n";
}

__host__ void Scene::loadShader() {
    std::string vertexCode, fragmentCode;
    std::ifstream vShaderFile, fShaderFile;

    vShaderFile.exceptions(std::ifstream::failbit | std::ifstream::badbit);
    fShaderFile.exceptions(std::ifstream::failbit | std::ifstream::badbit);
    try {

        vShaderFile.open(vertexShaderPath);
        fShaderFile.open(fragmentShaderPath);

        std::stringstream vShaderStream, fShaderStream;

        vShaderStream << vShaderFile.rdbuf();
        fShaderStream << fShaderFile.rdbuf();

        vShaderFile.close();
        fShaderFile.close();

        vertexCode = vShaderStream.str();
        fragmentCode = fShaderStream.str();
    } catch(std::ifstream::failure &e) {
        std::cerr << "ERROR::SHADER::FILE_NOT_SUCCESSFULLY_READ: " << e.what() << std::endl;
        std::cout << fragmentShaderPath << " " << vertexShaderPath << '\n';
        throw std::runtime_error("Shader File not readable");
    }
    const char *vShaderCode = vertexCode.c_str();
    const char *fShaderCode = fragmentCode.c_str();


    unsigned int vertex = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertex, 1, &vShaderCode, nullptr);
    glCompileShader(vertex);
    checkShaderCompileError(vertex, "VERTEX");

    unsigned int fragment = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragment, 1, &fShaderCode, nullptr);
    glCompileShader(fragment);
    checkShaderCompileError(fragment, "FRAGMENT");

    shaderID = glCreateProgram();
    glAttachShader(shaderID, vertex);
    glAttachShader(shaderID, fragment);
    glLinkProgram(shaderID);
    checkShaderCompileError(shaderID, "PROGRAM");

    glDeleteShader(vertex);
    glDeleteShader(fragment);
}

__host__ void Scene::checkShaderCompileError(unsigned int shader, const std::string &type) {
    GLint success;
    GLchar infoLog[1024];
    if(type != "PROGRAM") {
        glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
        if(!success) {
            glGetShaderInfoLog(shader, 1024, nullptr, infoLog);
            std::cerr << "ERROR::SHADER_COMPILATION_ERROR of type: " << type << "\n"
                      << infoLog
                      << "\n -- --------------------------------------------------- -- " << std::endl;
        }
    } else {
        glGetProgramiv(shader, GL_LINK_STATUS, &success);
        if(!success) {
            glGetProgramInfoLog(shader, 1024, NULL, infoLog);
            std::cerr << "ERROR::PROGRAM_LINKING_ERROR of type: " << type << "\n"
                      << infoLog
                      << "\n -- --------------------------------------------------- -- " << std::endl;
        }
    }
}
