
#include <hip/hip_runtime.h>
////
//// Created by steinraf on 20/08/22.
////
//
//#include "material.h"
//#include "utility/warp.h"
//
//
//__device__ bool Lambertian::scatter(const Ray &rayIn, const HitRecord &rec, Vector3f &attenuation, Ray &scattered,
//                                    Sampler &sampler) const {
//
//    Vector3f scatter = rec.normal + Warp::RandomInUnitSphere(sampler);
//    if (scatter.squaredNorm() < 1e-5)
//        scatter = rec.normal;
//
//    const Vector3f target = rec.position + scatter;
//
//    scattered = Ray(rec.position, target - rec.position);
//    attenuation = albedo;
//
//    return true;
//}