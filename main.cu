#include <filesystem>


#include "src/utility/vector.h"
#include "src/scene/scene.h"
#include "src/scene/sceneLoader.h"


int main(int argc, char **argv){
    std::cout << "Parsing obj...\n";

//    const std::filesystem::path filePath = "./scenes/simple.xml";
    const std::filesystem::path filePath = "./scenes/clocks.xml";


    assert(filePath.extension() == ".xml");

    std::cout << "Starting rendering...\n";

    Scene s(SceneRepresentation(filePath), Device::CPU);

    s.renderCPU();
//    s.renderGPU();




    std::cout << "Drew image to file\n";


//    hipDeviceReset();

    return EXIT_SUCCESS;
}
